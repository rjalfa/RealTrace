#include "hip/hip_runtime.h"
#include "kernel.h"
#include "structures.h"
#include <thrust/reduce.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include "camera.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime_api.h"
#include "utilities.h"
#include <queue>

#define TX 32
#define TY 32
#define AMBIENT_COLOR make_float3(0.235294, 0.67451, 0.843137)
#define KR 0.001
#define KT 0.1
#define EULER_CONSTANT 2.718
#define eta 4.0
#define KA 0.4

__device__ unsigned char clip(float x) { return x > 255 ? 255 : (x < 0 ? 0 : x); }

BVHTree * d_tree;
float3* colors = 0;

Ray* d_rays[7];
float* d_coeffs[7];
float** d_d_coeffs = NULL;
hipEvent_t event;
hipStream_t streamA1, streamA2, streamA3, streamA4;


// kernel function to compute decay and shading
__device__ void get_color_from_float3(float3 color, uchar4* cell)
{
	cell->x = clip(color.x * 255);
	cell->y = clip(color.y * 255);
	cell->z = clip(color.z * 255);
	cell->w = 255;
}

__device__ bool refract(const float3& I, const float3& N, const float e, float3& T)
{
	float k = 1.0 - e * e * (1.0 - dotProduct(N, I) * dotProduct(N, I));
	if (k < 0) return false;
	T = e * I - (e * dotProduct(N, I) + sqrt(k)) * N;
	return true;
}

__device__ void fresnel(const float3& I, const float3& N, const float& ior, float &kr)
{
	float cosi = clamp(-1, 1, dotProduct(I, N));
	float etai = 1, etat = ior;
	if (cosi > 0) {
		float t = etai;
		etai = etat;
		etat = t;
	}
	// Compute sini using Snell's law
	float sint = etai / etat * sqrtf(max(0.f, 1 - cosi * cosi));
	// Total internal reflection
	if (sint >= 1) {
		kr = 1;
	}
	else {
		float cost = sqrtf(max(0.f, 1 - sint * sint));
		cosi = fabsf(cosi);
		float Rs = ((etat * cosi) - (etai * cost)) / ((etat * cosi) + (etai * cost));
		float Rp = ((etai * cosi) - (etat * cost)) / ((etai * cosi) + (etat * cost));
		kr = (Rs * Rs + Rp * Rp) / 2;
	}
}

//Shared Memory Loop Intersect

__device__ void intersect(Triangle* triangles, int num_triangles, Ray* r, BVHTree * root)
{
//  __shared__ Triangle localObjects[32];
//  int triangles_to_scan = num_triangles;
//  while(triangles_to_scan > 0)
//  {
//    int x = min(triangles_to_scan,32);
//    if(threadIdx.x == 0 && threadIdx.y < x) localObjects[threadIdx.y] = triangles[threadIdx.y];
//    __syncthreads();
//
//    for(int i = 0; i < x; i ++) localObjects[i].intersect(r);
//    triangles += 32;
//    triangles_to_scan -= 32;
//    __syncthreads();
//  }
	root->intersect(triangles, *r, 0);
}

////////////////////////////////////////////////////////////////////////////
// Ray generation kernel
// Parameters:
// camera = Camera object
// w = width
// h = height
// out_rays = Output rays
// d_out = Output image to be resetted
////////////////////////////////////////////////////////////////////////////
__global__ void createRaysAndResetImage(Camera* camera, int w, int h, Ray* out_rays, uchar4* d_out, float* d_coeffs[7], float3* out_color)
{
	if (!camera || !out_rays || !d_out) return;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	float3 pos = camera->get_position();
	float3 dir = camera->get_ray_direction(i, j);
	int index = i + j * w; // 1D indexing
	out_rays[index] = Ray(pos, dir);
	d_out[index] = make_uchar4(0, 0, 0, 0);

	out_color[index] = make_float3(0, 0, 0);
	for (int i = 1; i < 7; i ++)
	{
		if (d_coeffs[i] != NULL) d_coeffs[i][index] = 0.0f;
	}
}

////////////////////////////////////////////////////////////////////////////
// Recursive Ray-tracing Kernel
// Parameters:
// out_color = Global Color Array that stores output from all kernels
// in_coeffs = The coeffs for the current kernel rays. If NULL, assumed all 1's
// w = width
// h = height
// rays = The rays to trace for this kernel
// out_rays_reflect = The rays that emerge from reflection from this kernel, If NULL, assumed end of recursion
// out_rays_refract = The rays that emerge from reflection from this kernel, If NULL, assumed end of recursion
// out_coeffs_reflect = The coeffs for the reflected rays
// out_coeffs_refract = The coeffs for the refracted rays
// triangles = Triangle objects
// num_triangles = Number of triangles in above
// l = LightSource object
// ug = UniformGrid object
////////////////////////////////////////////////////////////////////////////
__global__ void raytrace(float3 *out_color, float* in_coeffs, int w, int h, Ray* rays, Ray* out_rays_reflect, float* out_coeffs_reflect, Ray* out_rays_refract, float* out_coeffs_refract, Triangle* triangles, int num_triangles, LightSource* l, BVHTree * root)
{
	if (out_color == NULL || rays == NULL) return;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int index = i + j * w;
	//Switches
	bool in_coeff = (in_coeffs != NULL) ? in_coeffs[index] : 1.00;

	if (in_coeff < EPSILON || rays[index].direction == make_float3(0, 0, 0)) return;

	bool can_refract = (out_rays_refract != NULL && out_coeffs_refract != NULL);
	bool can_reflect = (out_rays_reflect != NULL && out_coeffs_reflect != NULL);
	bool will_refract = false;
	bool will_reflect = false;
	//Get owned ray
	Ray ray = rays[index];
	intersect(triangles, num_triangles, &ray, root);
	//bool reflect_over_refract = false;
	//Do one time intersection
	float3 finalColor = make_float3(0, 0, 0);
	if (!ray.has_intersected) finalColor = AMBIENT_COLOR;
	else
	{
		float3 I = normalize(ray.direction);
		float3 N = normalize(ray.intersected->get_normal());
		will_reflect = (ray.intersected->type_of_material == REFLECTIVE);
		will_refract = (ray.intersected->type_of_material == REFRACTIVE);
		finalColor = get_light_color(get_point(&ray, ray.t), N, l, ray.intersected, I);
		finalColor = finalColor + (ray.intersected)->color * KA;
		if ((!can_reflect && !can_refract) || (!will_reflect && !will_refract)) {  }
		//Reflect
		else if (can_reflect && will_reflect)
		{
			float3 R = reflect(I, N);
			Ray reflectedRay(ray.getPosition() + 1e-4 * R, R);
			out_rays_reflect[index] = reflectedRay;
			out_coeffs_reflect[index] = in_coeff * KR;
			finalColor = finalColor * (1 - KR);
		}
		else if (can_refract && will_refract)
		{
			/*
			float c = 0;//,k = 0;
			float3 R = reflect(I,N);
			float3 T;
			if(dotProduct(I,N) < 0)
			{
			  refract(I,N,eta,T);
			  c = -dotProduct(I,N);
			}
			else
			{
			  //k = 1;
			  //k = make_float3(pow(EULER_CONSTANT,-1.0*0.27*t),pow(EULER_CONSTANT,-1.0*0.45*t),pow(EULER_CONSTANT,-1.0*0.55*t));
			  if(refract(I,-1.0*N,1/eta,T)) c = dotProduct(T,N);
			  else {
				Ray reflectedRay(ray.getPosition()+ 1e-4 * R,R);
				out_rays_reflect[index] = reflectedRay;
				out_coeffs_reflect[index] = in_coeff * KR;
				finalColor = finalColor * (1-KR);
				//return k*shade_ray(temp);
				reflect_over_refract = true;
			  }
			}
			if(!reflect_over_refract)
			{
			  float _R0 = ((eta-1)*(eta-1))/((eta+1)*(eta+1));
			  float _R = _R0 + (1-_R0)*pow(1-c,5);
			  Ray temp1 = Ray(ray.getPosition()+ 1e-4 * R,R);
			  Ray temp2 = Ray(ray.getPosition()+ 1e-4 * T,T);
			  out_rays_reflect[index] = temp1;
			  out_coeffs_reflect[index] = in_coeff * _R;
			  out_rays_refract[index] = temp2;
			  out_coeffs_refract[index] = in_coeff * (1-_R);
			  in_coeff = 0;
			}
			*/
			float3 refractionColor = make_float3(0, 0, 0);
			// compute fresnel
			float kr;
			float3 hitPoint = ray.getPosition();
			fresnel(I, N, eta, kr);
			bool outside = (dotProduct(I, N) < 0);
			float3 bias = N * 1e-4f;
			// compute refraction if it is not a case of total internal reflection
			if (kr < 1) {
				float3 refractionDirection;
				refract(I, N, eta, refractionDirection);
				refractionDirection = normalize(refractionDirection);
				float3 refractionRayOrig = outside ? hitPoint - bias : hitPoint + bias;
				Ray refractedRay(refractionRayOrig, refractionDirection);
				out_rays_refract[index] = refractedRay;
				out_coeffs_refract[index] = in_coeff * (1 - kr);
				//refractionColor = castRay(refractionRayOrig, refractionDirection, objects, lights, options, depth + 1);
			}
			float3 reflectionDirection = normalize(reflect(I, N));
			float3 reflectionRayOrig = outside ? hitPoint + bias : hitPoint - bias;
			//float3 reflectionColor = castRay(reflectionRayOrig, reflectionDirection, objects, lights, options, depth + 1);

			out_rays_refract[index] = Ray(reflectionRayOrig, reflectionDirection);
			out_coeffs_refract[index] = in_coeff * kr;

			// mix the two
			//finalColor += reflectionColor * kr + refractionColor * (1 - kr);
			in_coeff = 0.0;
		}

	}
	finalColor = finalColor * in_coeff;
	// out_color[index] = finalColor;
	atomicAdd(&out_color[index].x, finalColor.x);
	atomicAdd(&out_color[index].y, finalColor.y);
	atomicAdd(&out_color[index].z, finalColor.z);
};

/*
Color World::shade_ray(Ray ray)
{
  if(ray.getLevel() > RECURSION_DEPTH) return background;
  firstIntersection(ray);
  if(ray.didHit())
  {
	// cout << ray.getDirection() << " " << ray.getIdx() << endl;
	// cerr << ray.getOrigin() << " " << ray.getDirection() << " " << ray.didHit() << endl;
	Color shadowColor(0.0,0.0,0.0);
	bool isShadow = false;
	//Run Shadow Test
	const Object* intersectedObject = ray.intersected();
	for(LightSource* ls : this->lightSourceList) {
	  Ray shadowRay(ray.getPosition()+0.01*(ls->getPosition()-ray.getPosition()),ls->getPosition()-ray.getPosition());
	  firstIntersection(shadowRay);
	  if(shadowRay.didHit()) {
		isShadow = true;
		shadowColor = ambient*(intersectedObject->getMaterial()->shade(ray))*(intersectedObject->getMaterial())->ka;
	  }
	}
	//..Compute Shade factor due to light
	Color lightColor(0.0,0.0,0.0);
	for(LightSource* ls : this->lightSourceList) {
	  // cerr << ray.getOrigin() << " " << ray.getDirection() << " " << ray.didHit() << " ";
	  // cerr << intersectedObject << endl;
	  lightColor = lightColor + get_light_shade(ray.getPosition(),intersectedObject->getNormalAtPosition(ray.getPosition()),*ls,intersectedObject->getMaterial(),ray.getDirection());
	}
	lightColor = lightColor + ambient*(intersectedObject->getMaterial()->shade(ray))*(intersectedObject->getMaterial())->ka;
	//if(shadowEffect) lightColor = lightColor*intersectedObject->getMaterial()->ka;

	Color finalColor = lightColor;
	if(isShadow) finalColor = finalColor*(1e-4) + shadowColor*(1 - 1e-4);

	//Reflection
	auto N = intersectedObject->getNormalAtPosition(ray.getPosition());
	auto I = ray.getDirection();
	N.normalize();
	I.normalize();

	double eta = intersectedObject->getMaterial()->eta;
	Vector3D T(0.0,0.0,0.0);
	double t = ray.getParameter();
	double c = 0;
	Vector3D k(1.0,1.0,1.0);
	int level = ray.getLevel();
	if(intersectedObject->getMaterial()->kr > 0 && intersectedObject->getMaterial()->kt > 0)
	{
	  //Dielectrics
	  auto R = reflect(I,N);
	  if(dotProduct(ray.getDirection(),N) < 0)
	  {
		refract(I,N,eta,T);
		c = -dotProduct(I,N);
	  }
	  else
	  {
		k = Vector3D(pow(EULER_CONSTANT,-1.0*0.27*t),pow(EULER_CONSTANT,-1.0*0.45*t),pow(EULER_CONSTANT,-1.0*0.55*t));
		if(refract(I,-1.0*N,1/eta,T)) c = dotProduct(T,N);
		else {
		  Ray temp = Ray(ray.getPosition()+ 1e-4 * R,R,level+1);
		  return k*shade_ray(temp);
		}
	  }
	  double _R0 = ((eta-1)*(eta-1))/((eta+1)*(eta+1));
	  double _R = _R0 + (1-_R0)*pow(1-c,5);
	  Ray temp1 = Ray(ray.getPosition()+ 1e-4 * R,R,level+1);
	  Ray temp2 = Ray(ray.getPosition()+ 1e-4 * T,T,level*2);
	  return k*(_R * shade_ray(temp1) + (1-_R)*shade_ray(temp2));
	}
	else if(intersectedObject->getMaterial()->kr > 0)
	{
	  auto R = reflect(I,N);
	  Ray reflectedRay(ray.getPosition()+ 1e-4 * R,R, level + 1);

	  finalColor = finalColor + (intersectedObject->getMaterial()->kr)*shade_ray(reflectedRay);
	}
	return finalColor;
  }
  return background;
}
*/
///////////////////////////////////////////////////////////////////
// Convert to RGBA kernel
// Parameters:
// color = Color array in floats
// d_out = Output array as RGBA unsigned char
// w = width
// h = height
//////////////////////////////////////////////////////////////////
__global__ void convert_to_rgba(float3 *color, uchar4* d_out, int w, int h)
{
	if (!color || !d_out) return ;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	int index = i + j * w; // 1D indexing
	get_color_from_float3(color[index], d_out + index);
}

int damnCeil(int num, int den) {
	return (num / den) + (num % den != 0);
}

__global__ void get_bounds(float * xmin, float * xmax, float * ymin, float * ymax, float * zmin, float * zmax, Triangle * triangles, int num_triangles) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < num_triangles) {
		triangles[idx].getWorldBound(xmin[idx], xmax[idx], ymin[idx], ymax[idx], zmin[idx], zmax[idx]);
	}
}

// Expands a 10-bit integer into 30 bits
// by inserting 2 zeros after each bit.
__device__ unsigned int expandBits(unsigned int v) {
    v = (v * 0x00010001u) & 0xFF0000FFu;
    v = (v * 0x00000101u) & 0x0F00F00Fu;
    v = (v * 0x00000011u) & 0xC30C30C3u;
    v = (v * 0x00000005u) & 0x49249249u;
    return v;
}

// Calculates a 30-bit Morton code for the
// given 3D point located within the unit cube [0,1].
__device__ unsigned int morton3D(float x, float y, float z, BBox * bounds) {
	x = (x - bounds->axis_min[0]) / (bounds->axis_max[0] - bounds->axis_min[0]);
	y = (y - bounds->axis_min[1]) / (bounds->axis_max[1] - bounds->axis_min[1]);
	z = (z - bounds->axis_min[2]) / (bounds->axis_max[2] - bounds->axis_min[2]);
    x = min(max(x * 1024.0f, 0.0f), 1023.0f);
    y = min(max(y * 1024.0f, 0.0f), 1023.0f);
    z = min(max(z * 1024.0f, 0.0f), 1023.0f);
    unsigned int xx = expandBits((unsigned int)x);
    unsigned int yy = expandBits((unsigned int)y);
    unsigned int zz = expandBits((unsigned int)z);
    return xx * 4 + yy * 2 + zz;
}

__global__ void generate_morton_codes(unsigned int * morton_codes, float * xmin, float * xmax, float * ymin,
                                float * ymax, float * zmin, float * zmax, BBox * bounds, int num_triangles) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < num_triangles) {
        morton_codes[idx] = morton3D(xmin[idx] + xmax[idx] / 2,
                                ymin[idx] + ymax[idx] / 2,
                                zmin[idx] + zmax[idx] / 2, bounds);
    }
}

int findSplit(unsigned int * sorted_codes, int first, int last) {
    // Identical Morton codes => split the range in the middle.

    unsigned int firstCode = sorted_codes[first];
    unsigned int lastCode = sorted_codes[last];

    if (firstCode == lastCode)
        return (first + last) >> 1;

    // Calculate the number of highest bits that are the same
    // for all objects, using the count-leading-zeros intrinsic.

//    int commonPrefix = __clz(firstCode ^ lastCode);
    int commonPrefix = __builtin_clz(firstCode ^ lastCode);
    // Use binary search to find where the next bit differs.
    // Specifically, we are looking for the highest object that
    // shares more than commonPrefix bits with the first one.

    int split = first; // initial guess
    int step = last - first;

    do {
        step = (step + 1) >> 1; // exponential decrease
        int newSplit = split + step; // proposed new position

        if (newSplit < last) {
            unsigned int splitCode = sorted_codes[newSplit];
//            int splitPrefix = __clz(firstCode ^ splitCode);
            int splitPrefix = __builtin_clz(firstCode ^ splitCode);
            if (splitPrefix > commonPrefix)
                split = newSplit; // accept proposal
        }
    } while (step > 1);

    return split;
}

void generateHierarchy(unsigned int * sorted_codes, int first, int last, Triangle * triangles, BVHTree& tree, int& idx) {
//	int curr_idx = idx;
//	if(first == last) {
//		tree.isLeaf[curr_idx] = true;
//		tree.primitive_idx[curr_idx] = first;
//		tree.left[curr_idx] = - 1;
//		tree.right[curr_idx] = -1;
//		tree.bbox[curr_idx] = triangles[first].getWorldBound();
//		return;
//    }
//
//	int split = findSplit(sorted_codes, first, last);
//    int left_idx = ++idx;
//    generateHierarchy(sorted_codes, first, split, triangles, tree, idx);
//    int right_idx = ++idx;
//    generateHierarchy(sorted_codes, split + 1, last, triangles, tree, idx);
//    tree.left[curr_idx] = left_idx, tree.right[curr_idx] = right_idx;
//    tree.isLeaf[curr_idx] = false;
//    tree.primitive_idx[curr_idx] = -1;
//    tree.bbox[curr_idx] = tree.bbox[left_idx];
//    tree.bbox[curr_idx].doUnion(tree.bbox[right_idx]);
//    return;

	queue < int > first_queue, last_queue, node_queue;
	node_queue.push(idx); idx++;
	first_queue.push(first);
	last_queue.push(last);
	while(!node_queue.empty()) {
		int node = node_queue.front(); node_queue.pop();
		int node_first = first_queue.front(); first_queue.pop();
		int node_last = last_queue.front(); last_queue.pop();
		if(node_first != node_last) {
			tree.isLeaf[node] = false;
			tree.primitive_idx[node] = -1;
			int split = findSplit(sorted_codes, node_first, node_last);
			tree.left[node] = idx;
			tree.right[node] = idx + 1;
			node_queue.push(idx); node_queue.push(idx + 1);
			idx += 2;
			first_queue.push(node_first); first_queue.push(split + 1);
			last_queue.push(split); last_queue.push(node_last);
		} else {
			tree.isLeaf[node] = true;
			tree.primitive_idx[node] = node_first;
			tree.left[node] = - 1;
			tree.right[node] = -1;
			tree.bbox[node] = triangles[node_first].getWorldBound();
		}
	}

	for(int i = idx - 1; i >= 0; i--) {
		if(!tree.isLeaf[i]) {
			tree.bbox[i] = tree.bbox[tree.left[i]];
			tree.bbox[i].doUnion(tree.bbox[tree.right[i]]);
		}
	}
}

void buildTree(int w, int h, Triangle * triangles, int num_triangles) {

    //checkCudaErrors(hipMalloc((void**)&colors, sizeof(float3)*w*h));
    //checkCudaErrors(hipMalloc((void**)&d_rays[0], sizeof(Ray)*w*h));

    float * xmin, * xmax, * ymin, * ymax, * zmin, * zmax;
    hipMalloc(&xmin, sizeof(float) * num_triangles);
    hipMalloc(&xmax, sizeof(float) * num_triangles);
    hipMalloc(&ymin, sizeof(float) * num_triangles);
    hipMalloc(&ymax, sizeof(float) * num_triangles);
    hipMalloc(&zmin, sizeof(float) * num_triangles);
    hipMalloc(&zmax, sizeof(float) * num_triangles);

    const dim3 blockSize(TX * TY);
    const dim3 gridSizeTriangles(damnCeil(num_triangles, TX * TY));

    get_bounds <<< gridSizeTriangles, blockSize >>> (xmin, xmax, ymin, ymax, zmin, zmax, triangles, num_triangles);

    thrust::tuple <float, float, float> axis_min, axis_max;

    thrust::device_ptr < float > xminptr = thrust::device_pointer_cast(xmin);
    thrust::device_ptr < float > xmaxptr = thrust::device_pointer_cast(xmax);
    thrust::device_ptr < float > yminptr = thrust::device_pointer_cast(ymin);
    thrust::device_ptr < float > ymaxptr = thrust::device_pointer_cast(ymax);
    thrust::device_ptr < float > zminptr = thrust::device_pointer_cast(zmin);
    thrust::device_ptr < float > zmaxptr = thrust::device_pointer_cast(zmax);

    BBox bounds, * d_bounds;
    checkCudaErrors(hipMalloc(&d_bounds, sizeof(BBox)));
    bounds.axis_min[0] = thrust::reduce(xminptr, xminptr + num_triangles, 1e36, thrust::minimum<float>());
    bounds.axis_min[1] = thrust::reduce(yminptr, yminptr + num_triangles, 1e36, thrust::minimum<float>());
    bounds.axis_min[2] = thrust::reduce(zminptr, zminptr + num_triangles, 1e36, thrust::minimum<float>());
    bounds.axis_max[0] = thrust::reduce(xmaxptr, xmaxptr + num_triangles, -1e36, thrust::maximum<float>());
    bounds.axis_max[1] = thrust::reduce(ymaxptr, ymaxptr + num_triangles, -1e36, thrust::maximum<float>());
    bounds.axis_max[2] = thrust::reduce(zmaxptr, zmaxptr + num_triangles, -1e36, thrust::maximum<float>());
    cout << bounds.axis_min[0] << " " << bounds.axis_max[0] << endl;
	cout << bounds.axis_min[1] << " " << bounds.axis_max[1] << endl;
	cout << bounds.axis_min[2] << " " << bounds.axis_max[2] << endl;
    checkCudaErrors(hipMemcpy(d_bounds, &bounds, sizeof(BBox), hipMemcpyHostToDevice));
//    hipDeviceSynchronize();
//    printf("%f %f\n", bounds.axis_min[0], bounds.axis_max[0]);
//    printf("%f %f\n", bounds.axis_min[1], bounds.axis_max[1]);
//    printf("%f %f\n", bounds.axis_min[2], bounds.axis_max[2]);
//    checkCudaErrors(hipMemcpy(&bounds, d_bounds, sizeof(BBox), hipMemcpyDeviceToHost));
//    printf("%f %f\n", bounds.axis_min[0], bounds.axis_max[0]);
//	printf("%f %f\n", bounds.axis_min[1], bounds.axis_max[1]);
//	printf("%f %f\n", bounds.axis_min[2], bounds.axis_max[2]);
    unsigned int * d_morton_codes;
    checkCudaErrors(hipMalloc(&d_morton_codes, sizeof(unsigned int) * num_triangles));
    hipDeviceSynchronize();
    cout << "allocation done" << endl;
    generate_morton_codes <<< gridSizeTriangles, blockSize >>> (d_morton_codes, xmin, xmax, ymin, ymax, zmin,
                                                                zmax, d_bounds, num_triangles);
    unsigned int * morton_codes = (unsigned int *) malloc(sizeof(unsigned int) * num_triangles);
    Triangle * h_triangles = (Triangle *) malloc(sizeof(Triangle) * num_triangles);

    checkCudaErrors(hipMemcpy(morton_codes, d_morton_codes, sizeof(unsigned int) * num_triangles, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_triangles, triangles, sizeof(Triangle) * num_triangles, hipMemcpyDeviceToHost));

    hipDeviceSynchronize();
    cout << "morton codes done" << endl;

    // thrust sort and stuff begin
    thrust::sort_by_key(morton_codes, morton_codes + num_triangles, h_triangles);
    // thrust sort and stuff done
    hipDeviceSynchronize();
    cout << "sort stuff done" << endl;

    int idx = 0;
    BVHTree h_tree(3 * num_triangles), h_dtree_holder;

    generateHierarchy(morton_codes, 0, num_triangles - 1, h_triangles, h_tree, idx);
    cout << num_triangles << " "  << idx << " " << (float) idx / num_triangles << endl;
    checkCudaErrors(hipMalloc(&h_dtree_holder.bbox, sizeof(BBox) * num_triangles * 3));
    checkCudaErrors(hipMalloc(&h_dtree_holder.left, sizeof(int)  * num_triangles * 3));
    checkCudaErrors(hipMalloc(&h_dtree_holder.right, sizeof(int) * num_triangles * 3));
    checkCudaErrors(hipMalloc(&h_dtree_holder.primitive_idx, sizeof(int) * num_triangles * 3));
    checkCudaErrors(hipMalloc(&h_dtree_holder.isLeaf, sizeof(bool) * num_triangles * 3));

    checkCudaErrors(hipMemcpy(h_dtree_holder.bbox, h_tree.bbox, sizeof(BBox) * num_triangles * 3, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(h_dtree_holder.left, h_tree.left, sizeof(int) * num_triangles * 3, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(h_dtree_holder.right, h_tree.right, sizeof(int) * num_triangles * 3, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(h_dtree_holder.primitive_idx, h_tree.primitive_idx, sizeof(int) * num_triangles * 3, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(h_dtree_holder.isLeaf, h_tree.isLeaf, sizeof(bool) * num_triangles * 3, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&d_tree, sizeof(BVHTree)));
    checkCudaErrors(hipMemcpy(d_tree, &h_dtree_holder, sizeof(BVHTree), hipMemcpyHostToDevice));
    hipDeviceSynchronize();
    cout << "all memory copy done" << endl;
    checkCudaErrors(hipFree(xmin));
    checkCudaErrors(hipFree(xmax));
    checkCudaErrors(hipFree(ymin));
    checkCudaErrors(hipFree(ymax));
    checkCudaErrors(hipFree(zmin));
    checkCudaErrors(hipFree(zmax));
}

void create_space_for_kernels(int w, int h)
{

	checkCudaErrors(hipMalloc((void**)&colors, sizeof(float3)*w * h));
	//checkCudaErrors(hipMalloc((void**)&d_rays[0], sizeof(Ray)*w*h));
	for (int i = 0; i < 7; i ++)
	{
		checkCudaErrors(hipMalloc((void**)&d_rays[i], sizeof(Ray)*w * h));
		if (i) checkCudaErrors(hipMalloc((void**)&d_coeffs[i], sizeof(float)*w * h));
	}

	hipEventCreate(&event);
	hipStreamCreate(&streamA1);
	hipStreamCreate(&streamA2);
	hipStreamCreate(&streamA3);
	hipStreamCreate(&streamA4);

	d_coeffs[0] = NULL;
	checkCudaErrors(hipMalloc((void**)&d_d_coeffs, sizeof(float*) * 7));
	checkCudaErrors(hipMemcpy(d_d_coeffs, d_coeffs, sizeof(float*) * 7, hipMemcpyHostToDevice));
}

void free_space_for_kernels()
{
	//if(colors) checkCudaErrors(hipFree(colors));
	for (int i = 0; i < 7; i ++)
	{
		checkCudaErrors(hipFree(d_rays[i]));
		if (i && d_coeffs[i]) checkCudaErrors(hipFree(d_coeffs[i]));
	}

	hipStreamDestroy(streamA1);
	hipStreamDestroy(streamA2);
	hipStreamDestroy(streamA3);
	hipStreamDestroy(streamA4);
	hipEventDestroy(event);

	checkCudaErrors(hipFree(d_d_coeffs));
}

void kernelLauncher(uchar4 *d_out, int w, int h, Camera* camera, Triangle* triangles, int num_triangles, LightSource* l) {
	const dim3 blockSize(TX, TY);
	const dim3 gridSize = dim3(w / TX, h / TY);

	//Start Procedure
	hipProfilerStart();

	createRaysAndResetImage <<< gridSize, blockSize>>>(camera, w, h, d_rays[0], d_out, d_d_coeffs, colors);
	hipDeviceSynchronize();

	//Karlo Ray trace 1000 baar yahaan
	//A
	raytrace <<< gridSize, blockSize, 0, streamA1>>>(colors, d_coeffs[0], w, h, d_rays[0], d_rays[1], d_coeffs[1], d_rays[2], d_coeffs[2], triangles, num_triangles, l, d_tree);
	//hipEventRecord(event);
	hipDeviceSynchronize();

	//Run these 2 concurrently
	//A1
	raytrace <<< gridSize, blockSize, 0, streamA1>>>(colors, d_coeffs[1], w, h, d_rays[1], d_rays[3], d_coeffs[3], d_rays[4], d_coeffs[4], triangles, num_triangles, l, d_tree);
	//A2
	raytrace <<< gridSize, blockSize, 0, streamA2>>>(colors, d_coeffs[2], w, h, d_rays[2], d_rays[5], d_coeffs[5], d_rays[6], d_coeffs[6], triangles, num_triangles, l, d_tree);
	//hipEventRecord(event);
	hipDeviceSynchronize();

	//Run these 4 concurrently
	//A11
	raytrace <<< gridSize, blockSize, 0, streamA1>>>(colors, d_coeffs[3], w, h, d_rays[3], NULL, NULL, NULL, NULL, triangles, num_triangles, l, d_tree);
	//A12
	raytrace <<< gridSize, blockSize, 0, streamA2>>>(colors, d_coeffs[4], w, h, d_rays[4], NULL, NULL, NULL, NULL, triangles, num_triangles, l, d_tree);
	//A21
	raytrace <<< gridSize, blockSize, 0, streamA3>>>(colors, d_coeffs[5], w, h, d_rays[5], NULL, NULL, NULL, NULL, triangles, num_triangles, l, d_tree);
	//A22
	raytrace <<< gridSize, blockSize, 0, streamA4>>>(colors, d_coeffs[6], w, h, d_rays[6], NULL, NULL, NULL, NULL, triangles, num_triangles, l, d_tree);
	//hipEventRecord(event);
	hipDeviceSynchronize();

	//Final Output Array
	convert_to_rgba <<< gridSize, blockSize>>>(colors, d_out, w, h);
	hipDeviceSynchronize();
	hipProfilerStop();
}
