#include "hip/hip_runtime.h"
#include "kernel.h"
#include "structures.h"
#include <thrust/reduce.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/scan.h>
#include "camera.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime_api.h"
#include "utilities.h"
#define AMBIENT_COLOR make_float3(0.235294, 0.67451, 0.843137)
#define KR 0.3
#define KT 0.3
#define EULER_CONSTANT 2.718
#define ETA 1.5
#define KA 0.4

__device__ inline unsigned char clip(float x) { return x > 255 ? 255 : (x < 0 ? 0 : x); }

UniformGrid * d_uniform_grid;
float3* colors = 0;

Ray* d_rays[7];
float* d_coeffs[7];
float** d_d_coeffs = NULL;
const int MAXSTREAMS = 4;
hipEvent_t event[MAXSTREAMS];
hipStream_t stream[MAXSTREAMS];

// kernel function to compute decay and shading
__device__ inline void get_color_from_float3(float3 color, uchar4* cell)
{
	cell->x = clip(color.x * 255);
	cell->y = clip(color.y * 255);
	cell->z = clip(color.z * 255);
	cell->w = 255;
}

__device__ float3 refract(const float3 &I, const float3 &N, const float &ior) 
{ 
	float cosi = clamp(-1, 1, dotProduct(I, N)); 
	float etai = 1, etat = ior; 
	float3 n = N; 
	if (cosi < 0) { cosi = -cosi; } else { 
		float temp = etai;
		etai = etat;
		etat = temp;
		//swap(etai, etat); 
		n= -N;
	} 
	float eta = etai / etat; 
	float k = 1 - eta * eta * (1 - cosi * cosi); 
	return (k < 0) ? make_float3(0,0,0) : (eta * I + (eta * cosi - __fsqrt_rz(k)) * n); 
}

__device__ __forceinline__ void fresnel(const float3& I, const float3& N, const float& ior, float &kr)
{
	float cosi = clamp(-1, 1, dotProduct(I, N));
	float etai = 1, etat = ior;
	if (cosi > 0) {
		float t = etai;
		etai = etat;
		etat = t;
	}
	// Compute sini using Snell's law
	float sint = etai / etat * __fsqrt_rz(max(0.f, 1 - cosi * cosi));
	// Total internal reflection
	if (sint >= 1) {
		kr = 1;
	}
	else {
		float cost = __fsqrt_rz(fmaxf(0.f, 1 - sint * sint));
		cosi = fabsf(cosi);
		float Rs = ((etat * cosi) - (etai * cost)) / ((etat * cosi) + (etai * cost));
		float Rp = ((etai * cosi) - (etat * cost)) / ((etai * cosi) + (etat * cost));
		kr = (Rs * Rs + Rp * Rp) / 2;
	}
}

//Uniform Grid Intersect
//__device__ void getFirstIntersection(UniformGrid * ug, Ray& r) {
//  ug->intersect(r);
//}

//Global Memory loop intersect
/*
__device__ void intersect(Triangle* triangles, int num_triangles, Ray* r)
{
  for(int i = 0; i < num_triangles; i ++) triangles[i].intersect(r);
}
*/

//Shared Memory Loop Intersect

__device__ inline void intersect(Triangle* triangles, int num_triangles, Ray* r, UniformGrid * ug, float in_coeff)
{
//  __shared__ Triangle localObjects[32];
//  int triangles_to_scan = num_triangles;
//  while(triangles_to_scan > 0)
//  {
//    int x = min(triangles_to_scan,32);
//    if(threadIdx.x == 0 && threadIdx.y < x) localObjects[threadIdx.y] = triangles[threadIdx.y];
//    __syncthreads();
//
//    for(int i = 0; i < x; i ++) localObjects[i].intersect(r);
//    triangles += 32;
//    triangles_to_scan -= 32;
//    __syncthreads();
//  }
	ug->intersect(triangles, *r, in_coeff);
}

////////////////////////////////////////////////////////////////////////////
// Ray generation kernel
// Parameters:
// camera = Camera object
// w = width
// h = height
// out_rays = Output rays
// d_out = Output image to be resetted
////////////////////////////////////////////////////////////////////////////
__global__ void createRaysAndResetImage(Camera* camera, int w, int h, Ray* out_rays, uchar4* d_out, float* d_coeffs[7], float3* out_color)
{
	if (!camera || !out_rays || !d_out) return;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	float3 pos = camera->get_position();
	float3 dir = camera->get_ray_direction(i, j);
	int index = i + j * w; // 1D indexing
	out_rays[index] = Ray(pos, dir);
	d_out[index] = make_uchar4(0, 0, 0, 0);
	out_color[index] = make_float3(0, 0, 0);
	for(int i = 0; i < 7; i ++) if(d_coeffs[i] != NULL) d_coeffs[i][index] = 0;
}

////////////////////////////////////////////////////////////////////////////
// Recursive Ray-tracing Kernel
// Parameters:
// out_color = Global Color Array that stores output from all kernels
// in_coeffs = The coeffs for the current kernel rays. If NULL, assumed all 1's
// w = width
// h = height
// rays = The rays to trace for this kernel
// out_rays_reflect = The rays that emerge from reflection from this kernel, If NULL, assumed end of recursion
// out_rays_refract = The rays that emerge from reflection from this kernel, If NULL, assumed end of recursion
// out_coeffs_reflect = The coeffs for the reflected rays
// out_coeffs_refract = The coeffs for the refracted rays
// triangles = Triangle objects
// num_triangles = Number of triangles in above
// l = LightSource object
// ug = UniformGrid object
////////////////////////////////////////////////////////////////////////////
__global__ void raytrace(float3 *out_color, float* in_coeffs, int w, int h, Ray* rays, Ray* out_rays_reflect, float* out_coeffs_reflect, Ray* out_rays_refract, float* out_coeffs_refract, Triangle* triangles, int num_triangles, LightSource* p_l, UniformGrid * p_ug)
{

	__shared__ LightSource l;
	__shared__ UniformGrid ug;

	if(threadIdx.x == 0 && threadIdx.y == 0) {l = *p_l; ug = *p_ug;}
	__syncthreads();

	if (out_color == NULL || rays == NULL) return;
	//int i = blockDim.x * blockIdx.x + threadIdx.x;
	//int j = blockDim.y * blockIdx.y + threadIdx.y;
	register int index = (blockDim.x * blockIdx.x + threadIdx.x) + (blockDim.y * blockIdx.y + threadIdx.y) * w;
	//Switches
	register float in_coeff = ((in_coeffs != NULL) ? in_coeffs[index] : 1.00);
	in_coeff = __saturatef(in_coeff);
	//clamp(in_coeff, 0, 1);
	Ray ray = rays[index];

	int flag = 0;
	flag |= (out_rays_refract != NULL && out_coeffs_refract != NULL);
	flag |= (out_rays_reflect != NULL && out_coeffs_reflect != NULL) << 1;
	if(out_coeffs_reflect != NULL) out_coeffs_reflect[index] = 0;
	if(out_coeffs_refract != NULL) out_coeffs_refract[index] = 0;

	//Get owned ray
	
	intersect(triangles, num_triangles, &ray, &ug, in_coeff);
	if (in_coeff < EPSILON || ray.direction == make_float3(0, 0, 0)) return;
	//bool reflect_over_refract = false;
	//Do one time intersection
	float3 finalColor = AMBIENT_COLOR;
	register Triangle* intersected = 0;
	if (ray.has_intersected) {
		intersected = ray.intersected;
		float3 I = normalize(ray.direction);
		float3 N = normalize(intersected->get_normal());
		flag |= (intersected->type_of_material == REFLECTIVE) << 2;
		flag |= (intersected->type_of_material == REFRACTIVE) << 3;
		finalColor = get_light_color(get_point(&ray, ray.t), N, &l, intersected, I);
		finalColor += (intersected)->color * KA;
		//Reflect
		float kr;
		bool outside = (dotProduct(I, N) < 0);
		float eff_eta = ETA;
		fresnel(I, N, eff_eta, kr);
		float3 bias = N * 1e-4f;
		float3 hitPoint = ray.getPosition();
		
		if(flag & 4) kr = KR;

		if(flag & 6 == 6 || ((flag & 1) && (flag & 8)))
		{
			float3 reflectionDirection = reflect(I, N);
			float3 reflectionRayOrig = outside ? hitPoint + bias : hitPoint - bias;
			out_rays_reflect[index] = Ray(reflectionRayOrig, reflectionDirection);
			out_coeffs_reflect[index] = in_coeff * kr;
			in_coeff *= (1 - kr);
		}
		
		if ((flag & 1) && (flag & 8))
		{
			if (kr < 1) {
				float3 refractionDirection = refract(I, N, eff_eta);
				float3 refractionRayOrig = outside ? hitPoint - bias : hitPoint + bias;
				out_rays_refract[index] = Ray(refractionRayOrig, refractionDirection);
				out_coeffs_refract[index] = in_coeff * (1 - kr);
			}
			in_coeff = 0.0;
		}
	}

	finalColor *= in_coeff;
	atomicAdd(&out_color[index].x, finalColor.x);
	atomicAdd(&out_color[index].y, finalColor.y);
	atomicAdd(&out_color[index].z, finalColor.z);
};

///////////////////////////////////////////////////////////////////
// Convert to RGBA kernel
// Parameters:
// color = Color array in floats
// d_out = Output array as RGBA unsigned char
// w = width
// h = height
//////////////////////////////////////////////////////////////////
__global__ void convert_to_rgba(float3 *color, uchar4* d_out, int w, int h)
{
	if (!color || !d_out) return ;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	int index = i + j * w; // 1D indexing
	get_color_from_float3(color[index], d_out + index);
}

int damnCeil(int num, int den) {
	return (num / den) + (num % den != 0);
}

__global__ void get_bounds(float * xmin, float * xmax, float * ymin, float * ymax, float * zmin, float * zmax, Triangle * triangles, int num_triangles) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < num_triangles) {
		triangles[idx].getWorldBound(xmin[idx], xmax[idx], ymin[idx], ymax[idx], zmin[idx], zmax[idx]);
	}
}

__global__ void count_sizes(UniformGrid * ug, Triangle * triangles, int num_triangles) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < num_triangles) {
		float xmin, xmax, ymin, ymax, zmin, zmax;
		triangles[idx].getWorldBound(xmin, xmax, ymin, ymax, zmin, zmax);

		int vxmin, vxmax, vymin, vymax, vzmin, vzmax;

		vxmin = ug->posToVoxel(xmin, 0);
		vxmax = ug->posToVoxel(xmax, 0);
		vymin = ug->posToVoxel(ymin, 1);
		vymax = ug->posToVoxel(ymax, 1);
		vzmin = ug->posToVoxel(zmin, 2);
		vzmax = ug->posToVoxel(zmax, 2);

		for (int z = vzmin; z <= vzmax; z++) {
			for (int y = vymin; y <= vymax; y++) {
				for (int x = vxmin; x <= vxmax; x++) {
					int o = ug->offset(x, y, z);
					atomicAdd(&(ug->lower_limit[o]), 1);
				}
			}
		}
	}
}

//__global__ void reserve_space(UniformGrid * ug, int nv) {
//  int idx = blockDim.x * blockIdx.x + threadIdx.x;
//  if(idx < nv) {
//    if(idx > 0) ug->voxels[idx].offset = ug->voxel_sizes[idx - 1];
//    else ug->voxels[idx].offset = 0;
//    if(idx > 0) ug->voxels[idx].max_size = ug->voxel_sizes[idx] - ug->voxel_sizes[idx - 1];
//    else ug->voxels[idx].max_size = ug->voxel_sizes[idx];
//    ug->voxels[idx].curr_size = 0;
////    printf("%d\n", ug->voxels[idx].max_size);
//  }
//}

__global__ void build_grid(UniformGrid * ug, Triangle * triangles, int num_triangles) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < num_triangles) {
		float xmin, xmax, ymin, ymax, zmin, zmax;
		triangles[idx].getWorldBound(xmin, xmax, ymin, ymax, zmin, zmax);

		int vxmin, vxmax, vymin, vymax, vzmin, vzmax;

		vxmin = ug->posToVoxel(xmin, 0);
		vxmax = ug->posToVoxel(xmax, 0);
		vymin = ug->posToVoxel(ymin, 1);
		vymax = ug->posToVoxel(ymax, 1);
		vzmin = ug->posToVoxel(zmin, 2);
		vzmax = ug->posToVoxel(zmax, 2);

		for (int z = vzmin; z <= vzmax; z++) {
			for (int y = vymin; y <= vymax; y++) {
				for (int x = vxmin; x <= vxmax; x++) {
					int o = ug->offset(x, y, z);
					Voxel::addPrimitive(ug, idx, o);
				}
			}
		}
	}
}

class justMax {
public:
	__host__ __device__
	thrust::tuple<float, float, float> operator()(thrust::tuple<float, float, float> a, thrust::tuple<float, float, float> b) {
		return thrust::make_tuple(max(thrust::get < 0 > (a), thrust::get < 0 > (b)), max(thrust::get < 1 > (a), thrust::get < 1 > (b)), max(thrust::get < 2 > (a), thrust::get < 2 > (b)));
	}
};

class justMin {
public:
	__host__ __device__
	thrust::tuple<float, float, float> operator()(thrust::tuple<float, float, float> a, thrust::tuple<float, float, float> b) {
		return thrust::make_tuple(min(thrust::get < 0 > (a), thrust::get < 0 > (b)), min(thrust::get < 1 > (a), thrust::get < 1 > (b)), min(thrust::get < 2 > (a), thrust::get < 2 > (b)));
	}
};

void buildGrid(int w, int h, Triangle * triangles, int num_triangles) {

	//checkCudaErrors(hipMalloc((void**)&colors, sizeof(float3)*w*h));
	//checkCudaErrors(hipMalloc((void**)&d_rays[0], sizeof(Ray)*w*h));

	float * xmin, * xmax, * ymin, * ymax, * zmin, * zmax;
	hipMalloc(&xmin, sizeof(float) * num_triangles);
	hipMalloc(&xmax, sizeof(float) * num_triangles);
	hipMalloc(&ymin, sizeof(float) * num_triangles);
	hipMalloc(&ymax, sizeof(float) * num_triangles);
	hipMalloc(&zmin, sizeof(float) * num_triangles);
	hipMalloc(&zmax, sizeof(float) * num_triangles);

	const dim3 blockSize(TX * TY);
	const dim3 gridSizeTriangles(damnCeil(num_triangles, TX * TY));

	get_bounds <<< gridSizeTriangles, blockSize >>> (xmin, xmax, ymin, ymax, zmin, zmax, triangles, num_triangles);

	thrust::tuple <float, float, float> axis_min, axis_max;

	thrust::device_ptr < float > xminptr = thrust::device_pointer_cast(xmin);
	thrust::device_ptr < float > xmaxptr = thrust::device_pointer_cast(xmax);
	thrust::device_ptr < float > yminptr = thrust::device_pointer_cast(ymin);
	thrust::device_ptr < float > ymaxptr = thrust::device_pointer_cast(ymax);
	thrust::device_ptr < float > zminptr = thrust::device_pointer_cast(zmin);
	thrust::device_ptr < float > zmaxptr = thrust::device_pointer_cast(zmax);

	UniformGrid h_uniform_grid;

	h_uniform_grid.bounds.axis_min[0] = thrust::reduce(xminptr, xminptr + num_triangles, 1e36, thrust::minimum<float>());
	h_uniform_grid.bounds.axis_min[1] = thrust::reduce(yminptr, yminptr + num_triangles, 1e36, thrust::minimum<float>());
	h_uniform_grid.bounds.axis_min[2] = thrust::reduce(zminptr, zminptr + num_triangles, 1e36, thrust::minimum<float>());
	h_uniform_grid.bounds.axis_max[0] = thrust::reduce(xmaxptr, xmaxptr + num_triangles, -1e36, thrust::maximum<float>());
	h_uniform_grid.bounds.axis_max[1] = thrust::reduce(ymaxptr, ymaxptr + num_triangles, -1e36, thrust::maximum<float>());
	h_uniform_grid.bounds.axis_max[2] = thrust::reduce(zmaxptr, zmaxptr + num_triangles, -1e36, thrust::maximum<float>());

	h_uniform_grid.initialize(num_triangles);

	checkCudaErrors(hipMalloc(&d_uniform_grid, sizeof(UniformGrid)));
	checkCudaErrors(hipMemcpy(d_uniform_grid, &h_uniform_grid, sizeof(UniformGrid), hipMemcpyHostToDevice));

	const dim3 gridSizeVoxels(damnCeil(h_uniform_grid.nv, TX * TY));
	count_sizes <<< gridSizeTriangles, blockSize >>> (d_uniform_grid, triangles, num_triangles);

	checkCudaErrors(hipMemcpy(&h_uniform_grid, d_uniform_grid, sizeof(UniformGrid), hipMemcpyDeviceToHost));

	thrust::device_ptr < int > voxel_sizes = thrust::device_pointer_cast(h_uniform_grid.lower_limit);
	int total_space = thrust::reduce(voxel_sizes, voxel_sizes + h_uniform_grid.nv);
	checkCudaErrors(hipMalloc(&(h_uniform_grid.index_pool), sizeof(int) * total_space));
	thrust::exclusive_scan(voxel_sizes, voxel_sizes + h_uniform_grid.nv, voxel_sizes);

	checkCudaErrors(hipMemcpy(h_uniform_grid.upper_limit, h_uniform_grid.lower_limit,
							   sizeof(int) * h_uniform_grid.nv, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(d_uniform_grid, &h_uniform_grid, sizeof(UniformGrid), hipMemcpyHostToDevice));

//  reserve_space <<< gridSizeVoxels, blockSize >>> (d_uniform_grid, h_uniform_grid.nv);

	build_grid <<< gridSizeTriangles, blockSize >>> (d_uniform_grid, triangles, num_triangles);

	checkCudaErrors(hipFree(xmin));
	checkCudaErrors(hipFree(xmax));
	checkCudaErrors(hipFree(ymin));
	checkCudaErrors(hipFree(ymax));
	checkCudaErrors(hipFree(zmin));
	checkCudaErrors(hipFree(zmax));
}

void create_space_for_kernels(int w, int h)
{

	checkCudaErrors(hipMalloc((void**)&colors, sizeof(float3)*w * h));
	//checkCudaErrors(hipMalloc((void**)&d_rays[0], sizeof(Ray)*w*h));
	for (int i = 0; i < 7; i ++) {
		checkCudaErrors(hipMalloc((void**)&d_rays[i], sizeof(Ray)*w * h));
		if(i) checkCudaErrors(hipMalloc((void**)&d_coeffs[i], sizeof(float)*w * h));
	}
	for(int i = 0; i < MAXSTREAMS; i++) {
		hipEventCreate(&event[i]);
		hipStreamCreate(&stream[i]);
	}

	d_coeffs[0] = NULL;
	checkCudaErrors(hipMalloc((void**)&d_d_coeffs, sizeof(float*) * 7));
	checkCudaErrors(hipMemcpy(d_d_coeffs, d_coeffs, sizeof(float*) * 7, hipMemcpyHostToDevice));
}

void free_space_for_kernels()
{
	if(colors) checkCudaErrors(hipFree(colors));
	for (int i = 0; i < 7; i ++) {
		checkCudaErrors(hipFree(d_rays[i]));
		if(i && d_coeffs[i]) checkCudaErrors(hipFree(d_coeffs[i]));
	}
	for(int i = 0; i < MAXSTREAMS; i++) {
		hipEventDestroy(event[i]);
		hipStreamDestroy(stream[i]);
	}
	
	checkCudaErrors(hipFree(d_d_coeffs));
}

void kernelLauncher(uchar4 *d_out, int w, int h, Camera* camera, Triangle* triangles, int num_triangles, LightSource* l) {
	const dim3 blockSize(TX, TY);
	const dim3 gridSize = dim3(w / TX, h / TY);

	//Start Procedure
	// hipProfilerStart();

	createRaysAndResetImage <<< gridSize, blockSize>>>(camera, w, h, d_rays[0], d_out, d_d_coeffs, colors);
	// hipDeviceSynchronize();

	//Karlo Ray trace 1000 baar yahaan
	//A
	raytrace <<< gridSize, blockSize, 0, stream[0]>>>(colors, d_coeffs[0], w, h, d_rays[0], d_rays[1], d_coeffs[1], d_rays[2], d_coeffs[2], triangles, num_triangles, l, d_uniform_grid);
	hipEventRecord(event[0], stream[0]);
	//Run these 2 concurrently
	
	//A1
	raytrace <<< gridSize, blockSize, 0, stream[0]>>>(colors, d_coeffs[1], w, h, d_rays[1], d_rays[3], d_coeffs[3], d_rays[4], d_coeffs[4], triangles, num_triangles, l, d_uniform_grid);
	//A2
	hipStreamWaitEvent(stream[1], event[0], 0);
	raytrace <<< gridSize, blockSize, 0, stream[1]>>>(colors, d_coeffs[2], w, h, d_rays[2], d_rays[5], d_coeffs[5], d_rays[6], d_coeffs[6], triangles, num_triangles, l, d_uniform_grid);
	hipEventRecord(event[1], stream[0]);
	hipEventRecord(event[2], stream[1]);
	
	//Run these 4 concurrently
	//A11
	raytrace <<< gridSize, blockSize, 0, stream[0]>>>(colors, d_coeffs[3], w, h, d_rays[3], NULL, NULL, NULL, NULL, triangles, num_triangles, l, d_uniform_grid);
	//A12
	hipStreamWaitEvent(stream[2], event[1], 0);
	raytrace <<< gridSize, blockSize, 0, stream[2]>>>(colors, d_coeffs[4], w, h, d_rays[4], NULL, NULL, NULL, NULL, triangles, num_triangles, l, d_uniform_grid);
	//A21
	raytrace <<< gridSize, blockSize, 0, stream[1]>>>(colors, d_coeffs[5], w, h, d_rays[5], NULL, NULL, NULL, NULL, triangles, num_triangles, l, d_uniform_grid);
	//A22
	hipStreamWaitEvent(stream[3], event[2], 0);
	raytrace <<< gridSize, blockSize, 0, stream[3]>>>(colors, d_coeffs[6], w, h, d_rays[6], NULL, NULL, NULL, NULL, triangles, num_triangles, l, d_uniform_grid);

	//Final Output Array
	convert_to_rgba <<< gridSize, blockSize>>>(colors, d_out, w, h);
	hipEventSynchronize(event[1]);
	hipEventSynchronize(event[2]);
	// hipProfilerStop();
}
