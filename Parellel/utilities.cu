#include "hip/hip_runtime.h"
#include "utilities.h"

__device__ float determinant(float a,float b,float c,float d)
{
	// a b
	// c d
	return a*d - b*c;
}

__device__ float determinant(float3 col1, float3 col2, float3 col3)
{
	return col1.x * determinant(col2.y,col3.y,col2.z,col3.z) - col1.y * determinant(col2.x,col3.x,col2.z,col3.z) + col1.z * determinant(col2.x,col3.x,col2.y,col3.y);
}