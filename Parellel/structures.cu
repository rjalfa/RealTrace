#include "hip/hip_runtime.h"
#include "structures.h"
#include "utilities.h"

__device__ float3 Triangle::get_normal()
{
	return make_float3(0,0,0);
}

__device__ bool operator==(const float3& v1, const float3& v2)
{
	if (v1.x != v2.x) return false;
	if (v1.y != v2.y) return false;
	if (v1.z != v2.z) return false;
	return true;
}

__device__ bool operator!=(const float3& v1, const float3& v2)
{
	return !(v1==v2);   
}

__device__ float3 operator+(const float3& v1, const float3& v2)
{
	return make_float3(v1.x+v2.x, v1.y+v2.y, v1.z+v2.z);
}

__device__ float3 operator-(const float3& v1, const float3& v2)
{
	return make_float3(v1.x-v2.x, v1.y-v2.y, v1.z-v2.z);   
}

__device__ float3 operator/(const float3& v, float scalar)
{
	return make_float3(v.x/scalar, v.y/scalar, v.z/scalar);   
}

__device__ float3 operator*(const float3& v, float scalar)
{
	return make_float3(v.x*scalar, v.y*scalar, v.z*scalar);       
}

//Unary
__device__ float3 operator-(const float3& v)
{
	return make_float3(-v.x, -v.y, -v.z);
}

__device__ float3 operator*(float scalar, const float3& v)
{
	return v*scalar;       
}

__device__ float3 operator*(const float3& v, const float3& v1)
{
	return make_float3(v.x*v1.x, v.y*v1.y, v.z*v1.z);
}

__device__ float squaredlength(const float3& f)
{ 
	return (f.x*f.x + f.y*f.y + f.z*f.z);
}

__device__ float length(const float3& f)
{
	return sqrt(squaredlength(f));
}

__device__ float3 normalize(const float3& f)
{
	return f / length(f);
}

__device__ float3 unitVector(const float3& v)
{
	float len  = length(v);
	return v / len;
}

__device__ float3 get_point(Ray* r, float t)
{
	return r->origin + t*r->direction;
}

__device__ float dotProduct(const float3& v1, const float3& v2)
{ return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z; }

__device__ float tripleProduct(const float3& v1,const float3& v2,const float3& v3)
{
	return dotProduct(( crossProduct(v1, v2)), v3);   
}

__device__ float distance(const float3& v1, const float3& v2)
{
	return sqrt((v1.x-v2.x)*(v1.x-v2.x) + (v1.y-v2.y)*(v1.y-v2.y) + (v1.z-v2.z)*(v1.z-v2.z));
}

__device__ float3 reflect(const float3& I, const float3& N)
{
	return I - 2.0f*dotProduct(N,I)*N;
}

__device__ float3 crossProduct(const float3& v1, const float3& v2)
{
	float3 tmp;
	tmp.x = v1.y * v2.z - v1.z * v2.y;
	tmp.y = v1.z * v2.x - v1.x * v2.z;
	tmp.z = v1.x * v2.y - v1.y * v2.x;
	return tmp; 
}


__device__ bool Triangle::intersect(Ray *r)
{
	float A = determinant(vertexA-vertexB,vertexA-vertexC,r->direction);
	if(abs(A) < EPSILON) return false;
	float beta = determinant(vertexA-r->origin,vertexA-vertexC,r->direction) / A;
	float gamma = determinant(vertexA-vertexB,vertexA-r->origin,r->direction) / A;
	float t = determinant(vertexA-vertexB,vertexA-vertexC,vertexA-r->origin) / A;
	if(!(beta > 0.0 && gamma > 0.0 && beta+gamma < 1.0)) return false;
	if(!r->has_intersected)	{
		r->has_intersected = true;
		r->t = t;
	}
	else r->t = (r->t)>t?t:r->t;
	return true;
}


__device__ float3 get_light_color(float3 point, float3 normal, LightSource* l, Triangle* t, float3 viewVector)
{
	float3 vLightPosition = l->position;
	float3 n = normalize(normal);
	float3 r = normalize(reflect(-normalize(vLightPosition-point),n));
	float dist = distance(point,vLightPosition);
	//float fatt = 1.0 / (1.0 + 0.05*dist);
	float diffuse = max(dotProduct(n,normalize(vLightPosition)),0.0f);
	float specular = max(pow(dotProduct(normalize(viewVector),r),128),0.0);
	return 0.8*diffuse*(l->color)*(t->color) + 0.1*specular*(l->color);
}